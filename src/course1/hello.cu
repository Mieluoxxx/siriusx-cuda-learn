#include <hip/hip_runtime.h>
#include <iostream>

// CUDA内核函数：在GPU上执行的函数
__global__ void hello_world(void) {
    // 打印当前线程的块索引和线程索引
    printf("block idx:%d thread idx: %d\n", blockIdx.x, threadIdx.x);
    
    // 只有每个块的第一个线程（线程索引为0）才打印Hello world
    if (threadIdx.x == 0) {
        printf("GPU: Hello world!\n");
    }
}

int main(int argc, char **argv) {
    printf("CPU: Hello world!\n");
    
    // 启动CUDA内核：1个线程块，每个块有10个线程
    hello_world<<<1, 10>>>();
    
    // 等待GPU完成所有操作
    hipDeviceSynchronize();
    
    // 检查CUDA操作是否出错
    if (hipGetLastError() != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(hipGetLastError())
                  << std::endl;
        return 1;
    } else {
        std::cout << "GPU: Hello world finished!" << std::endl;
    }
    
    std::cout << "CPU: Hello world finished!" << std::endl;
    return 0;
}